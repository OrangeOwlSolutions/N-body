#include <thrust\host_vector.h>
#include <thrust\unique.h>
#include <thrust\merge.h>
//#include <thrust/execution_policy.h>

/********/
/* MAIN */
/********/
int main() {

	thrust::host_vector<int> h_a(6);
	thrust::host_vector<int> h_b(7);
	thrust::host_vector<int> h_c(13);
	h_a[0] = 1; h_a[1] = 3; h_a[2] = 5; h_a[3] = 7; h_a[4] = 9; h_a[5] = 11;
	h_b[0] = 1; h_b[1] = 1; h_b[2] = 2; h_b[3] = 3; h_b[4] = 5; h_b[5] = 8; h_b[6] = 13;

	thrust::merge(h_a.begin(), h_a.end(), h_b.begin(), h_b.end(), h_c.begin());

	//h_b.insert(h_b.end(), h_a.begin(), h_a.end());
	
	printf("After merging\n");
	for (int k = 0; k < 13; k++) printf("h_c[%d] = %d\n", k, h_c[k]);

	auto new_end = thrust::unique(h_c.begin(), h_c.end());
	
	int new_elements = new_end - h_c.begin();
	
	printf("\nAfter unique\n");
	for (int k = 0; k < new_elements; k++) printf("h_c[%d] = %d\n", k, h_c[k]);

	return 0;
}
